#include "hip/hip_runtime.h"


#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
//#include "pso-mult-swarm-sequencial.h"

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

typedef struct Particle
{
    double *position_i;
    double *velocity_i;
    double *pos_best_i;
    double err_best_i = -1;
    double err_i = -1;
} Particle;

typedef struct Swarm
{
    Particle *swarm;
    double *pos_best_s;
    double *err_best_s;
} Swarm;

const double MaxValue = 1.7976931348623157E+308;
const int num_dimensions = 2;
const int num_particle = 4096;
// const int num_swarms = 16;
const int THREAD_PER_BLOCK = 32;
const int BLOCKS = num_particle / THREAD_PER_BLOCK;
const int MAX_ITER = 30;

const double BOUNDS_SPHERE[] = {-10, 10, -10, 10};
double h_initial[] = {5, 5};
double *initial, *BOUNDS, *numbersRand;
double h_BOUNDS[] = {-10, 10, -10, 10};
Swarm *swarms, *h_swarms, *out_swarm;
double *pos_best_g, *h_pos_best_g;
double *err_best_g, h_err_best_g;
Particle *swarm, *h_swarm;

__device__ double gpuRandomNumberUniform(hiprandState_t state)
{
    hiprand_init(0, 0, 0, &state);
    return hiprand_uniform(&state);
}

__device__ double sphere(Particle *x)
{
    double total = 0.0;
    for (int i = 0; i <= num_dimensions; i++)
    {
        total += pow(x->position_i[i], 2);
    }
    return total;
}

__global__ void update_AllParticle_position_velocity(Swarm *swarms, double *bounds, double *pos_best_g, double *numbers, int num_swarms, int num_particle, int num_dimensions)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double w = 0.4;
    int c1 = 1;
    int c2 = 2;
    int c3 = 1;

    if (num_particle < i)
    {
        for (int k = 0; k < num_swarms; k++)
        {
            for (int j = 0; j < num_dimensions; j++)
            {
                double r1 = numbers[i];
                double r2 = numbers[i + j];
                double r3 = numbers[i];

                double vel_cognitive = c1 * r1 * (swarms[k].swarm[i].pos_best_i[j] - swarms[k].swarm[i].position_i[j]);
                double vel_social = c2 * r2 * (pos_best_g[j] - swarms[k].swarm[i].position_i[j]);
                double vel_sbest = c3 * r3 * (swarms[k].pos_best_s[j] - swarms[k].swarm[i].position_i[j]);
                swarms[k].swarm[i].velocity_i[j] = w * swarms[k].swarm[i].velocity_i[j] + vel_cognitive + vel_social + vel_sbest;

                swarms[k].swarm[j].position_i[i] = swarms[k].swarm[j].position_i[i] + swarms[k].swarm[j].velocity_i[i];

                if (swarms[k].swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
                {
                    swarms[k].swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
                }

                if (swarms[k].swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
                {
                    swarms[k].swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
                }
            }
        }
    }
}

__global__ void gpuGenerateRand(double *numbers)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state;
    hiprand_init(0, 0, i, &state);
    double number = hiprand_uniform(&state);
    numbers[i] = number;
}

__global__ void init_particle(Swarm *swarms, double *x0, double *numbers, int num_swarms)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    for (int k = 0.; k < num_swarms; k++)
    {
        for (int j = 0; j < num_dimensions; j++)
        {
            swarms[k].swarm[i].position_i[j] = x0[j];
            swarms[k].swarm[i].velocity_i[j] = numbers[i + j + k];
        }
    }
}

void initialize_gpu_memory(int num_swarms)
{
    h_pos_best_g = (double *)malloc(sizeof(double) * num_dimensions);
    h_pos_best_g[0] = MaxValue;
    h_pos_best_g[1] = MaxValue;
    h_err_best_g = MaxValue;

    h_swarm = (Particle *)malloc(sizeof(Particle) * num_particle);
    h_swarms = (Swarm *)malloc(sizeof(Swarm) * num_swarms);
    hipMalloc(&pos_best_g, sizeof(double) * num_dimensions);
    hipMalloc(&err_best_g, sizeof(double));
    hipMalloc(&swarm, sizeof(Particle) * num_particle);
    hipMalloc(&swarms, sizeof(Swarm) * num_swarms);
    hipMalloc(&BOUNDS, sizeof(double) * num_dimensions * num_dimensions);
    hipMalloc(&initial, sizeof(double) * num_dimensions);

    hipMalloc(&numbersRand, sizeof(double) * num_dimensions * num_particle * num_swarms);

    gpuGenerateRand<<<BLOCKS, THREAD_PER_BLOCK>>>(numbersRand);

    hipMemcpyAsync(BOUNDS, h_BOUNDS, sizeof(double) * num_dimensions * num_dimensions, hipMemcpyHostToDevice);
    hipMemcpyAsync(initial, h_initial, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);

    for (int k = 0; k < num_swarms; k++)
    {
        double h_err_best_s = MaxValue;
        HANDLE_ERROR(hipMalloc(&h_swarms[k].swarm, sizeof(Particle) * num_particle));
        HANDLE_ERROR(hipMalloc(&h_swarms[k].pos_best_s, sizeof(double) * num_dimensions));
        HANDLE_ERROR(hipMalloc(&h_swarms[k].err_best_s, sizeof(double)));

        for (int i = 0; i < num_particle; i++)
        {
            HANDLE_ERROR(hipMalloc(&h_swarm[i].position_i, sizeof(double) * num_dimensions));
            HANDLE_ERROR(hipMalloc(&h_swarm[i].velocity_i, sizeof(double) * num_dimensions));
            HANDLE_ERROR(hipMalloc(&h_swarm[i].pos_best_i, sizeof(double) * num_dimensions));
        }

        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].err_best_s, &h_err_best_s, sizeof(double), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].pos_best_s, &h_pos_best_g, sizeof(double) * num_dimensions, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].swarm, h_swarm, sizeof(Particle) * num_particle, hipMemcpyHostToDevice));
    }

    HANDLE_ERROR(hipMemcpy(swarms, h_swarms, sizeof(Swarm) * num_swarms, hipMemcpyHostToDevice));
}

__global__ void pso(Particle *swarm, double *pos_best_s, double *err_best_s, double *bounds, double *numbers)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    swarm[i].err_i = sphere(&swarm[i]);

    if (swarm[i].err_i < swarm[i].err_best_i)
    {
        swarm[i].pos_best_i = swarm[i].position_i;
        swarm[i].err_best_i = swarm[i].err_i;
    }

    if (swarm[i].err_i < *err_best_s || *err_best_s == -1)
    {
        pos_best_s[0] = swarm[i].position_i[0];
        pos_best_s[1] = swarm[i].position_i[1];
        *err_best_s = swarm[i].err_i;
    }

    double w = 0.9;
    int c1 = 1;
    int c2 = 2;

    for (int j = 0; j < num_dimensions; j++)
    {
        swarm[i].position_i[j] = swarm[i].position_i[j] + swarm[i].velocity_i[j];
        if (swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
        }

        if (swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
        }

        double r1 = numbers[i];
        double r2 = numbers[i + j];
        double vel_cognitive = c1 * r1 * (swarm[i].pos_best_i[j] - swarm[i].position_i[j]);
        double vel_social = c2 * r2 * (pos_best_s[j] - swarm[i].position_i[j]);
        swarm[i].velocity_i[j] = w * swarm[i].velocity_i[j] + vel_cognitive + vel_social;
    }
}

__global__ void calculate_fitness(Particle *swarm)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double err = sphere(&swarm[i]);
    swarm[i].err_i = err;
}

__global__ void evaluate_update_pbest(Particle *swarm)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (swarm[i].err_i < swarm[i].err_best_i)
    {
        swarm[i].pos_best_i = swarm[i].position_i;
        swarm[i].err_best_i = swarm[i].err_i;
    }
}

__global__ void update_gbest(Particle *swarm, double *pos_best_s, double *err_best_s)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (swarm[i].err_i < *err_best_s || *err_best_s == -1)
    {
        pos_best_s[0] = swarm[i].position_i[0];
        pos_best_s[1] = swarm[i].position_i[1];
        *err_best_s = swarm[i].err_i;
        // printf("Solution Swarm:%d [x:%.20f, y:% .20f]\n", i, swarm[i].position_i[0], swarm[i].position_i[1]);
    }
}

__global__ void update_position_velocity(Particle *swarm, double *bounds, double *pos_best_g, double *numbers)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double w = 0.9;
    int c1 = 1;
    int c2 = 2;

    for (int j = 0; j < num_dimensions; j++)
    {
        swarm[i].position_i[j] = swarm[i].position_i[j] + swarm[i].velocity_i[j];
        if (swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
        }

        if (swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
        }

        double r1 = numbers[i];
        double r2 = numbers[i + j];
        double vel_cognitive = c1 * r1 * (swarm[i].pos_best_i[j] - swarm[i].position_i[j]);
        double vel_social = c2 * r2 * (pos_best_g[j] - swarm[i].position_i[j]);
        swarm[i].velocity_i[j] = w * swarm[i].velocity_i[j] + vel_cognitive + vel_social;
    }
}

void pso_execute_stream(int num_swarms)
{

    int repeat = 1;
    float gtime = 0.0;
    for (int t = 0; t < repeat; t++)
    {

        hipDeviceReset();

        float time = 0.0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        initialize_gpu_memory(num_swarms);
        init_particle<<<BLOCKS, THREAD_PER_BLOCK>>>(swarms, initial, numbersRand, num_swarms);
        hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * num_swarms);

        double h_pos_best_s[2];
        double h_err_best_s;
        int k = 0;
        for (int i = 0.; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipStreamCreate(&stream[i]));
        }

        while (k < MAX_ITER)
        {
            for (int i = 0.; i < num_swarms; i++)
            {
                calculate_fitness<<<BLOCKS, THREAD_PER_BLOCK, 0, stream[i]>>>(h_swarms[i].swarm);
                evaluate_update_pbest<<<BLOCKS, THREAD_PER_BLOCK, 0, stream[i]>>>(h_swarms[i].swarm);
                update_gbest<<<BLOCKS, THREAD_PER_BLOCK, 0, stream[i]>>>(h_swarms[i].swarm, h_swarms[i].pos_best_s, h_swarms[i].err_best_s);
                update_position_velocity<<<BLOCKS, THREAD_PER_BLOCK, 0, stream[i]>>>(h_swarms[i].swarm, BOUNDS, h_swarms[i].pos_best_s, numbersRand);
            }
            k++;
        }

        for (int i = 0.; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipMemcpyAsync(h_pos_best_s, h_swarms[i].pos_best_s, sizeof(double) * num_dimensions, hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipMemcpyAsync(&h_err_best_s, h_swarms[i].err_best_s, sizeof(double), hipMemcpyDeviceToHost));
            if (h_pos_best_s[0] < h_pos_best_g[0] && h_pos_best_s[1] < h_pos_best_g[1])
            {
                h_pos_best_g[0] = h_pos_best_s[0];
                h_pos_best_g[1] = h_pos_best_s[1];
                h_err_best_g = h_err_best_s;
            }
            // printf("Solution Swarm:%d [x:%.20f, y:% .20f] error: % .20f\n", i, h_pos_best_s[0], h_pos_best_s[1], h_err_best_s);
        }

        update_AllParticle_position_velocity<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms, BOUNDS, pos_best_g, numbersRand, num_swarms, num_particle, num_dimensions);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        // printf("Final Solution: [x:%.20f, y:% .20f] error: % .20f\n", h_pos_best_g[0], h_pos_best_g[1], h_err_best_g);
        gtime += time;

        for (int i = 0; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipStreamDestroy(stream[i]));
        }

        hipError_t err = hipGetLastError();
        HANDLE_ERROR(err);
    }

    printf("Result Mult-Stream Swarms: %d Time: %3.2f ms.\n", num_swarms, gtime / repeat);
}

void pso_execute(int num_swarms)
{
    int repeat = 10;
    float gtime = 0.0;
    for (int t = 0; t < repeat; t++)
    {

        hipDeviceReset();

        h_pos_best_g = (double *)malloc(sizeof(double) * num_dimensions);
        h_pos_best_g[0] = MaxValue;
        h_pos_best_g[1] = MaxValue;
        h_err_best_g = MaxValue;

        h_swarm = (Particle *)malloc(sizeof(Particle) * num_particle);
        h_swarms = (Swarm *)malloc(sizeof(Swarm) * num_swarms);
        hipMalloc(&pos_best_g, sizeof(double) * num_dimensions);
        hipMalloc(&err_best_g, sizeof(double));
        hipMalloc(&swarm, sizeof(Particle) * num_particle);
        hipMalloc(&swarms, sizeof(Swarm) * num_swarms);
        hipMalloc(&BOUNDS, sizeof(double) * num_dimensions * num_dimensions);
        hipMalloc(&initial, sizeof(double) * num_dimensions);

        hipMalloc(&numbersRand, sizeof(double) * num_dimensions * num_particle * num_swarms);

        gpuGenerateRand<<<BLOCKS, THREAD_PER_BLOCK>>>(numbersRand);

        hipMemcpy(BOUNDS, h_BOUNDS, sizeof(double) * num_dimensions * num_dimensions, hipMemcpyHostToDevice);
        hipMemcpy(initial, h_initial, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);

        for (int k = 0; k < num_swarms; k++)
        {
            double h_err_best_s = MaxValue;
            HANDLE_ERROR(hipMalloc(&h_swarms[k].swarm, sizeof(Particle) * num_particle));
            HANDLE_ERROR(hipMalloc(&h_swarms[k].pos_best_s, sizeof(double) * num_dimensions));
            HANDLE_ERROR(hipMalloc(&h_swarms[k].err_best_s, sizeof(double)));

            for (int i = 0; i < num_particle; i++)
            {
                HANDLE_ERROR(hipMalloc(&h_swarm[i].position_i, sizeof(double) * num_dimensions));
                HANDLE_ERROR(hipMalloc(&h_swarm[i].velocity_i, sizeof(double) * num_dimensions));
                HANDLE_ERROR(hipMalloc(&h_swarm[i].pos_best_i, sizeof(double) * num_dimensions));
            }

            HANDLE_ERROR(hipMemcpy(h_swarms[k].err_best_s, &h_err_best_s, sizeof(double), hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(h_swarms[k].pos_best_s, &h_pos_best_g, sizeof(double) * num_dimensions, hipMemcpyHostToDevice));
            HANDLE_ERROR(hipMemcpy(h_swarms[k].swarm, h_swarm, sizeof(Particle) * num_particle, hipMemcpyHostToDevice));
        }

        HANDLE_ERROR(hipMemcpy(swarms, h_swarms, sizeof(Swarm) * num_swarms, hipMemcpyHostToDevice));

        float time = 0.0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        initialize_gpu_memory(num_swarms);
        init_particle<<<BLOCKS, THREAD_PER_BLOCK>>>(swarms, initial, numbersRand, num_swarms);

        double h_pos_best_s[2];
        double h_err_best_s;
        int k = 0;
        while (k < MAX_ITER)
        {
            for (int i = 0.; i < num_swarms; i++)
            {
                calculate_fitness<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms[i].swarm);
                evaluate_update_pbest<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms[i].swarm);
                update_gbest<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms[i].swarm, h_swarms[i].pos_best_s, h_swarms[i].err_best_s);
                update_position_velocity<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms[i].swarm, BOUNDS, h_swarms[i].pos_best_s, numbersRand);
            }
            k++;
        }

        for (int i = 0.; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipMemcpy(h_pos_best_s, h_swarms[i].pos_best_s, sizeof(double) * num_dimensions, hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipMemcpy(&h_err_best_s, h_swarms[i].err_best_s, sizeof(double), hipMemcpyDeviceToHost));

            if (h_pos_best_s[0] < h_pos_best_g[0] && h_pos_best_s[1] < h_pos_best_g[1])
            {
                h_pos_best_g[0] = h_pos_best_s[0];
                h_pos_best_g[1] = h_pos_best_s[1];
                h_err_best_g = h_err_best_s;
            }

            // printf("Solution Swarm:%d [x:%.20f, y:% .20f] error: % .20f\n", i, h_pos_best_s[0], h_pos_best_s[1], h_err_best_s);
        }

        update_AllParticle_position_velocity<<<BLOCKS, THREAD_PER_BLOCK>>>(h_swarms, BOUNDS, pos_best_g, numbersRand, num_swarms, num_particle, num_dimensions);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        // printf("Final Solution: [x:%.20f, y:% .20f] error: % .20f\n", h_pos_best_g[0], h_pos_best_g[1], h_err_best_g);
        gtime += time;

        hipError_t err = hipGetLastError();
        HANDLE_ERROR(err);
    }

    printf("Result Default Stream Swarms: %d Time: %3.2f ms.\n", num_swarms, gtime / repeat);
}

int main()
{
    /*pso_execute(2);
    pso_execute(4);
    pso_execute(8);
    pso_execute(16);
    pso_execute(32);
    pso_execute(64);*/
    hipProfilerStart();
    pso_execute_stream(2);
    /*pso_execute_stream(4);
    pso_execute_stream(8);
    pso_execute_stream(16);
    pso_execute_stream(32);
    pso_execute_stream(64);*/

    /*pso_execute_sequencial(2, 3000);
    pso_execute_sequencial(4, 3000);
    pso_execute_sequencial(8, 3000);
    pso_execute_sequencial(16, 3000);
    pso_execute_sequencial(32, 3000);
    pso_execute_sequencial(64, 3000);*/
    hipProfilerStop();
    return 0;
}
