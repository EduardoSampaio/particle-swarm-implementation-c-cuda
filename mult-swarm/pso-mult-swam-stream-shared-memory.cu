#include "hip/hip_runtime.h"


#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
//#include "pso-mult-swarm-sequencial.h"

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

typedef struct Particle
{
    double *position_i;
    double *velocity_i;
    double *pos_best_i;
    double err_best_i = -1;
    double err_i = -1;
} Particle;

typedef struct Swarm
{
    Particle *swarm;
    double *pos_best_s;
    double *err_best_s;
} Swarm;

const double MaxValue = 1.7976931348623157E+308;
const int num_dimensions = 2;
const int num_particle = 4096;
const int THREAD_PER_BLOCK = 128;
const int BLOCKS = num_particle / THREAD_PER_BLOCK;
const int MAX_ITER = 300;

double h_initial[] = {5, 5};
double *initial, *BOUNDS, *numbersRand;
double h_BOUNDS[] = {-10, 10, -10, 10};
Swarm *swarms, *h_swarms, *out_swarm;
double *pos_best_g, *h_pos_best_g;
double *err_best_g, h_err_best_g;
Particle *swarm, *h_swarm;

__device__ double gpuRandomNumberUniform(hiprandState_t state)
{
    hiprand_init(0, 0, 0, &state);
    return hiprand_uniform(&state);
}

__device__ double sphere(Particle *x)
{
    double total = 0.0;
    for (int i = 0; i <= num_dimensions; i++)
    {
        total += pow(x->position_i[i], 2);
    }
    return total;
}

__global__ void update_AllParticle_position_velocity(Swarm *swarms, double *bounds, double *pos_best_g, double *numbers, int num_swarms, int num_particle, int num_dimensions)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double w = 0.4;
    int c1 = 1;
    int c2 = 2;
    int c3 = 1;

    if (num_particle < i)
    {
        for (int k = 0; k < num_swarms; k++)
        {
            for (int j = 0; j < num_dimensions; j++)
            {
                double r1 = numbers[i];
                double r2 = numbers[i + j];
                double r3 = numbers[i];

                double vel_cognitive = c1 * r1 * (swarms[k].swarm[i].pos_best_i[j] - swarms[k].swarm[i].position_i[j]);
                double vel_social = c2 * r2 * (pos_best_g[j] - swarms[k].swarm[i].position_i[j]);
                double vel_sbest = c3 * r3 * (swarms[k].pos_best_s[j] - swarms[k].swarm[i].position_i[j]);
                swarms[k].swarm[i].velocity_i[j] = w * swarms[k].swarm[i].velocity_i[j] + vel_cognitive + vel_social + vel_sbest;

                swarms[k].swarm[j].position_i[i] = swarms[k].swarm[j].position_i[i] + swarms[k].swarm[j].velocity_i[i];

                if (swarms[k].swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
                {
                    swarms[k].swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
                }

                if (swarms[k].swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
                {
                    swarms[k].swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
                }
            }
        }
    }
}

__global__ void gpuGenerateRand(double *numbers)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state;
    hiprand_init(0, 0, i, &state);
    double number = hiprand_uniform(&state);
    numbers[i] = number;
}

__global__ void init_particle(Particle *swarm, double *x0, double *numbers, int)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    for (int j = 0; j < num_dimensions; j++)
    {
        swarm[i].position_i[j] = x0[j];
        swarm[i].velocity_i[j] = numbers[i + j];
    }
}

void initialize_gpu_memory(int num_swarms)
{
    h_pos_best_g = (double *)malloc(sizeof(double) * num_dimensions);
    h_pos_best_g[0] = MaxValue;
    h_pos_best_g[1] = MaxValue;
    h_err_best_g = MaxValue;

    h_swarm = (Particle *)malloc(sizeof(Particle) * num_particle);
    h_swarms = (Swarm *)malloc(sizeof(Swarm) * num_swarms);
    hipMalloc(&pos_best_g, sizeof(double) * num_dimensions);
    hipMalloc(&err_best_g, sizeof(double));
    hipMalloc(&BOUNDS, sizeof(double) * num_dimensions * num_dimensions);
    hipMalloc(&initial, sizeof(double) * num_dimensions);
    hipMalloc(&numbersRand, sizeof(double) * num_dimensions * num_particle * num_swarms);

    for (int k = 0; k < num_swarms; k++)
    {
        double h_err_best_s = MaxValue;
        HANDLE_ERROR(hipMalloc(&h_swarms[k].swarm, sizeof(Particle) * num_particle));
        HANDLE_ERROR(hipMalloc(&h_swarms[k].pos_best_s, sizeof(double) * num_dimensions));
        HANDLE_ERROR(hipMalloc(&h_swarms[k].err_best_s, sizeof(double)));

        for (int i = 0; i < num_particle; i++)
        {
            HANDLE_ERROR(hipMalloc(&h_swarm[i].position_i, sizeof(double) * num_dimensions));
            HANDLE_ERROR(hipMalloc(&h_swarm[i].velocity_i, sizeof(double) * num_dimensions));
            HANDLE_ERROR(hipMalloc(&h_swarm[i].pos_best_i, sizeof(double) * num_dimensions));
        }

        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].err_best_s, &h_err_best_s, sizeof(double), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].pos_best_s, &h_pos_best_g, sizeof(double) * num_dimensions, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpyAsync(h_swarms[k].swarm, h_swarm, sizeof(Particle) * num_particle, hipMemcpyHostToDevice));
    }
    hipMemcpyAsync(BOUNDS, h_BOUNDS, sizeof(double) * num_dimensions * num_dimensions, hipMemcpyHostToDevice);
    hipMemcpyAsync(initial, h_initial, sizeof(double) * num_dimensions, hipMemcpyHostToDevice);
    gpuGenerateRand<<<BLOCKS, THREAD_PER_BLOCK>>>(numbersRand);
}

__global__ void pso(Particle *swarm, double *pos_best_s, double *err_best_s, double *bounds, double *numbers, double *x0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ double shared_pos_best_s[2];
    __shared__ double shared_err_best_s[1];

    for (int j = 0; j < num_dimensions; j++)
    {
        swarm[i].position_i[j] = x0[j];
        swarm[i].velocity_i[j] = numbers[i + j];
    }

    __syncthreads();

    int k = 0;
    while (k < MAX_ITER)
    {

        swarm[i].err_i = sphere(&swarm[i]);

        if (swarm[i].err_i < swarm[i].err_best_i)
        {
            swarm[i].pos_best_i = swarm[i].position_i;
            swarm[i].err_best_i = swarm[i].err_i;
        }

        if (swarm[i].err_i < *err_best_s)
        {
            shared_pos_best_s[0] = swarm[i].position_i[0];
            shared_pos_best_s[1] = swarm[i].position_i[1];
            shared_err_best_s[0] = swarm[i].err_i;
        }

        double w = 0.9;
        int c1 = 1;
        int c2 = 2;

        for (int j = 0; j < num_dimensions; j++)
        {
            swarm[i].position_i[j] = swarm[i].position_i[j] + swarm[i].velocity_i[j];
            if (swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
            {
                swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
            }

            if (swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
            {
                swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
            }

            double r1 = numbers[i];
            double r2 = numbers[i + j];
            double vel_cognitive = c1 * r1 * (swarm[i].pos_best_i[j] - swarm[i].position_i[j]);
            double vel_social = c2 * r2 * (pos_best_s[j] - swarm[i].position_i[j]);
            swarm[i].velocity_i[j] = w * swarm[i].velocity_i[j] + vel_cognitive + vel_social;
        }
        __syncthreads();
        pos_best_s[0] = shared_pos_best_s[0];
        pos_best_s[1] = shared_pos_best_s[1];
        *err_best_s = shared_err_best_s[0];
        k++;
    }
}

__global__ void calculate_fitness(Particle *swarm)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double err = sphere(&swarm[i]);
    swarm[i].err_i = err;
}

__global__ void evaluate_update_pbest(Particle *swarm)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (swarm[i].err_i < swarm[i].err_best_i)
    {
        swarm[i].pos_best_i = swarm[i].position_i;
        swarm[i].err_best_i = swarm[i].err_i;
    }
}

__global__ void update_gbest(Particle *swarm, double *pos_best_s, double *err_best_s)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (swarm[i].err_i < *err_best_s || *err_best_s == -1)
    {
        pos_best_s[0] = swarm[i].position_i[0];
        pos_best_s[1] = swarm[i].position_i[1];
        *err_best_s = swarm[i].err_i;
    }
}

__global__ void update_position_velocity(Particle *swarm, double *bounds, double *pos_best_g, double *numbers)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double w = 0.9;
    int c1 = 1;
    int c2 = 2;

    for (int j = 0; j < num_dimensions; j++)
    {
        swarm[i].position_i[j] = swarm[i].position_i[j] + swarm[i].velocity_i[j];
        if (swarm[i].position_i[j] > bounds[j * num_dimensions + 1])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 1];
        }

        if (swarm[i].position_i[j] < bounds[j * num_dimensions + 0])
        {
            swarm[i].position_i[j] = bounds[j * num_dimensions + 0];
        }

        double r1 = numbers[i];
        double r2 = numbers[i + j];
        double vel_cognitive = c1 * r1 * (swarm[i].pos_best_i[j] - swarm[i].position_i[j]);
        double vel_social = c2 * r2 * (pos_best_g[j] - swarm[i].position_i[j]);
        swarm[i].velocity_i[j] = w * swarm[i].velocity_i[j] + vel_cognitive + vel_social;
    }
}

void pso_execute_stream(int num_swarms, int repeat)
{
    float gtime = 0.0;
    for (int t = 0; t < repeat; t++)
    {

        hipDeviceReset();

        float time = 0.0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        initialize_gpu_memory(num_swarms);
        hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t) * num_swarms);
        double h_pos_best_s[2];
        double h_err_best_s;

        for (int i = 0.; i < num_swarms; i++)
        {
            hipStreamCreate(&stream[i]);
            pso<<<BLOCKS, THREAD_PER_BLOCK, 0, stream[i]>>>(h_swarms[i].swarm, h_swarms[i].pos_best_s, h_swarms[i].err_best_s, BOUNDS, numbersRand, initial);
        }
        hipDeviceSynchronize();
        for (int i = 0.; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipMemcpyAsync(h_pos_best_s, h_swarms[i].pos_best_s, sizeof(double) * num_dimensions, hipMemcpyDeviceToHost));
            HANDLE_ERROR(hipMemcpyAsync(&h_err_best_s, h_swarms[i].err_best_s, sizeof(double), hipMemcpyDeviceToHost));
            if (h_pos_best_s[0] < h_pos_best_g[0] && h_pos_best_s[1] < h_pos_best_g[1])
            {
                h_pos_best_g[0] = h_pos_best_s[0];
                h_pos_best_g[1] = h_pos_best_s[1];
                h_err_best_g = h_err_best_s;
            }
        }
        printf("Final Solution: [x:%.20f, y:% .20f] error: % .20f\n", h_pos_best_g[0], h_pos_best_g[1], h_err_best_g);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);

        gtime += time;

        for (int i = 0; i < num_swarms; i++)
        {
            HANDLE_ERROR(hipStreamDestroy(stream[i]));
        }

        hipError_t err = hipGetLastError();
        HANDLE_ERROR(err);
    }

    printf("Result Mult-Stream Swarms: %d Time: %3.2f ms.\n", num_swarms, gtime / repeat);
}

int main()
{

    hipProfilerStart();
    pso_execute_stream(4, 1);
    hipProfilerStop();
    return 0;
}
